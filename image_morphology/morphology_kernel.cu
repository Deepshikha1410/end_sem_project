#include <hip/hip_runtime.h>
#include <algorithm>

__device__ unsigned char max(unsigned char a, unsigned char b) {
    return (a > b) ? a : b;
}

__device__ unsigned char min(unsigned char a, unsigned char b) {
    return (a < b) ? a : b;
}

__global__ void morphology_kernel(const unsigned char* input, unsigned char* output, int width, int height, int kernel_size, bool is_dilation) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        unsigned char result = (is_dilation) ? 0 : 255;

        int half_kernel = kernel_size / 2;

        for (int ky = -half_kernel; ky <= half_kernel; ++ky) {
            for (int kx = -half_kernel; kx <= half_kernel; ++kx) {
                int nx = min(max(x + kx, 0), width - 1);
                int ny = min(max(y + ky, 0), height - 1);
                unsigned char pixel = input[ny * width + nx];
                if (is_dilation) {
                    result = max(result, pixel);
                } else {
                    result = min(result, pixel);
                }
            }
        }

        output[y * width + x] = result;
    }
}

extern "C" void perform_morphology(const unsigned char* h_input, unsigned char* h_output, int width, int height, bool is_dilation) {
    unsigned char* d_input;
    unsigned char* d_output;
    int kernel_size = 3; // Example kernel size

    hipMalloc(&d_input, width * height * sizeof(unsigned char));
    hipMalloc(&d_output, width * height * sizeof(unsigned char));
    hipMemcpy(d_input, h_input, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    morphology_kernel<<<gridSize, blockSize>>>(d_input, d_output, width, height, kernel_size, is_dilation);

    hipMemcpy(h_output, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
